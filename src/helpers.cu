#include "hip/hip_runtime.h"
#include "helpers.h"


#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

__global__ void init_rand(int w, int h, float* weights) {
  int row = blockIdx.y * blockDim.y + threadIdx.y; 
  int column = blockIdx.x * blockDim.x + threadIdx.x; 
  if (row < h && column < w) {
    hiprandState state; // State for the random number generator
    hiprand_init(42, row * w + column, 0, &state); // Initialize the state
    weights[row * w + column] = sqrtf(2.0 / w) * hiprand_normal(&state);
  }
}

void print_matrix(int w, int h, float* matrix, std::string title)
{
  float* m_h = new float[w*h];
  hipMemcpy(m_h, matrix, w*h*sizeof(float), hipMemcpyDeviceToHost);
  std::cout<<title<<std::endl;
  for(int i = 0; i<h; i++)
  {
    for(int j = 0; j<w; j++)
    {
      std::cout<<std::fixed<<std::setprecision(3)<<m_h[i*w+j]<<", ";
    }
    std::cout<<std::endl;
  }
  free(m_h);
}

void initLayer(float* weights, float* biases, int w, int h, int BLOCK_SIZE)
{
  dim3 dimGrid = dim3(ceil(w/(float)BLOCK_SIZE), ceil(h/(float)BLOCK_SIZE), 1);
  dim3 dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
  init_rand<<<dimGrid, dimBlock>>>(w, h, weights);
  gpuErrchk(hipPeekAtLastError());

  dimGrid = dim3(ceil(h/(float)BLOCK_SIZE), 1, 1);
  dimBlock = dim3(BLOCK_SIZE, 1, 1);
  init_rand<<<dimGrid, dimBlock>>>(1, h, biases);
  gpuErrchk(hipPeekAtLastError());
}

void read_mnist(const std::string filename, int length, float* x, float* y)
{
  // std::cout << "DEBUG: reading " << filename << std::endl;
  int input_size = 784;
  int labels = 10;

  std::fstream fin;
  fin.open(filename);
  std::string row;
  constexpr char delim = ',';
  for(int i = 0; i<length; i++)
    {
      fin >> row;
      int pos = row.find(delim);
      if (pos == std::string::npos) {
	std::cout << "DEBUG: Processing row " << i << ": " << row << std::endl;
	std::cerr << "ERROR: Malformed CSV row (missing delimiter) at row " << i << std::endl;
	std::exit(1);
      }
      std::string label_str = row.substr(0, pos+1);
      int label;
      try {
	label = std::stoi(label_str);

	for(int j = 0; j < labels; j++)
	  {
	    y[labels * i + j] = (j == label);
	  }
      } catch (const std::exception& e) {
	std::cout << "DEBUG: Processing row " << i << ": " << row << std::endl;
	std::cout << "DEBUG: Extracted label string: " << label_str << std::endl;
	std::cerr << "ERROR: Failed to convert label to int at row " << i << ": " << e.what() << std::endl;
	std::exit(1);
      }

    for(int j = 0; j<labels; j++)
    {
      y[labels*i + j] = (j==label);
    }
    row.erase(0, pos+1);
    for(int j = 0; j<input_size; j++)
    {
      pos = row.find(delim);
      if (pos == std::string::npos)
      {
        pos = row.length() - 1;
      }
      x[i*input_size+j] = std::stof(row.substr(0, pos+1)) / 255; //normalize value
      row.erase(0, pos+1);
    }
    ASSERT(row.length() == 0, "didn't parse all values in row, %d", i);
  }
}

